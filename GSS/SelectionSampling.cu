#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <stdlib.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include <fstream>
#include <thrust/execution_policy.h>

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <thrust/scan.h>
#include <ctime>
using namespace std;
#define GPUCHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

//heapify subroutines.
__host__ __device__ inline int parent(int i)
{
	return i / 2;
}

__host__ __device__ inline int left(int i)
{
	return 2 * i + 1;
}

__host__ __device__ inline int right(int i)
{
	return 2 * i + 2;
}


template <typename T>
void viewGPUArray(T* array,  int num, string filename)
{
	const string dir = "C:/Users/lunamu/Dropbox/MATLAB/";
	T* host_array = new T[num]; 
	GPUCHECK(hipMemcpy(host_array, array, sizeof(T) * num, hipMemcpyDeviceToHost));
	ofstream file(dir + filename);
	for (int i = 0; i < num; i++)
	{
		file << host_array[i] << endl;
	}
}

template <typename T>
void viewGPUArrayMat(T* array, int num, string filename)
{
	const string dir = "C:/Users/lunamu/Dropbox/MATLAB/";
	T* host_array = new T[num];
	GPUCHECK(hipMemcpy(host_array, array, sizeof(T) * num, hipMemcpyDeviceToHost));
	ofstream file(dir + filename);
	for (int i = 0; i < num; i++)
	{
		file << host_array[i].p.x <<" "<<host_array[i].p.y << endl;
	}
}


//Code on testing hash grid on GPU
//1. Build a hash grid on GPU(Hash grid of a structure I defined)

//The class I'm using
__host__ __device__ inline float max_(const float &a, const float &b)
{
	return ((a > b) ? a : b);
}
__host__ __device__ inline float min_(const float &a, const float &b)
{
	return ((a < b) ? a : b);
}
__host__ __device__ class Point2D
{
	
public:
	float x, y, w;
	__host__ __device__ Point2D& operator=(const Point2D& target){ x = target.x; y = target.y; w = target.w; return *this; }
	__host__ __device__ Point2D operator+(const Point2D& b){ Point2D results; results.x = x + b.x; results.y = y + b.y; results.w = w + b.w; return results; }
	__host__ __device__ Point2D operator+(const float b)
	{ 
		Point2D results; 
		results.x = min_(1, x + b);
		results.y = min_(1, y + b);
		return results; 
	}
	__host__ __device__ Point2D operator-(const float b)
	{ 
		Point2D results; 
		results.x = max_(0,x - b); 
		results.y = max_(0,y - b); 
		return results; 
	}
	friend ostream& operator<<(ostream& os, const Point2D& p)
	{
		os << p.w ;
		return os;
	}
};
struct BBox
{
	float minx, miny;
	float maxx, maxy;
};

void bBox(vector<Point2D> points, BBox& bbox, size_t num)
{
	float minx = 1.0; float miny = 1.0;
	float maxx = 0.0; float maxy = 0.0;
	for (int i = 0; i < num; i++)
	{
		if (points[i].x < minx)minx = points[i].x;
		if (points[i].x > maxx)maxx = points[i].x;
		if (points[i].y < miny)miny = points[i].y;
		if (points[i].y > maxy)maxy = points[i].y;

	}
	bbox.minx = minx; bbox.maxx = maxx;
	bbox.miny = miny; bbox.maxy = maxy;
}

class HashValue
{
public:
	unsigned int morton;
	Point2D p;
	bool operator<(const HashValue& rhs) const {return  (morton < rhs.morton); }
	friend ostream& operator<<(ostream& os, const HashValue& h)
	{
		os << h.morton<<" "<<h.p;
		return os;
	}
};
__host__ __device__ void swap(HashValue& a, HashValue& b)
{
	HashValue tmp;
	tmp = a;
	a = b;
	b = tmp;
}



//Generate a million random points;

void generateRandomPointCloud(vector<Point2D>& points, size_t size = 1000000)
{
	//std::cout << "Generating " << size << " point cloud...";
	points.resize(size);
	for (size_t i = 0; i<size; i++)
	{
		
		points[i].x = (rand() % RAND_MAX) / float(RAND_MAX);
		points[i].y = (rand() % RAND_MAX) / float(RAND_MAX);
		points[i].w = 0.0;
	}

	//std::cout << "done\n";
}


//attention, grid_dim is represented by 2 to the power of grid_dim
//grid_dim less than 2^16 (for long int)
__host__ __device__ unsigned int mortonHash2D(Point2D point, size_t grid_dim)
{
	int x_axis = 0;
	int y_axis = 0;
	x_axis = (int)((point.x / 1.0) * (1<<grid_dim));
	y_axis = (int)((point.y / 1.0) * (1<<grid_dim));

	int interleaved_x = 0;
	int interleaved_y = 0;
	int mark = 0x01;
	for (int i = 0; i < grid_dim; i++)
	{
		interleaved_x |= ((x_axis & mark) << i << i);
		x_axis = x_axis >> 1;

		interleaved_y |= ((y_axis & mark) << i << i);
		y_axis = y_axis >> 1;
	}

	return (interleaved_x << 1) | (interleaved_y);
}
__host__ __device__ unsigned int mortonHash2D_axis(int x, int y, size_t grid_dim)
{
	int interleaved_x = 0;
	int interleaved_y = 0;
	int mark = 0x01;
	
	for (int i = 0; i < grid_dim; i++)
	{
		interleaved_x |= ((x & mark) << i << i);
		x = x >> 1;

		interleaved_y |= ((y & mark) << i << i);
		y = y >> 1;
	}

	return (interleaved_x << 1) | (interleaved_y);
}
struct HashElem
{
	int idx;
	size_t num;
	friend ostream& operator<<(ostream& os, const HashElem& h)
	{
		os << h.idx<<" "<<h.num;
		return os;
	}
};

__host__ __device__ void minHeapify(HashValue* A, int i, int size)
{
	/*int l = left(i);
	int r = right(i);
	int smallest;
	if ((l <= (size - 1)) && (A[l].p.w < A[i].p.w))
	{
		smallest = l;
	}
	else
	{
		smallest = i;
	}
	if ((r <= (size - 1)) && (A[r].p.w < A[smallest].p.w))
	{
		smallest = r;
	}
	if (smallest != i)
	{
		swap(A[i], A[smallest]);
		minHeapify(A, smallest, size);
	}
*/
	int cur = i;
	int l; int r;
	while (cur < size)
	{
		l = left(cur);
		r = right(cur);
		int smallest;
		if ((l >= size) || (r >= size))break;
		else
		{
			if ((l < size) && (A[l].p.w < A[i].p.w))
			{
				smallest = l;
			}
			else
			{
				smallest = cur;
			}
			if ((r < size) && (A[r].p.w < A[smallest].p.w))
			{
				smallest = r;
			}
			if (smallest != cur)
			{
				swap(A[i], A[smallest]);
				cur = smallest;
			}
			else
			{
				break;
			}
		}
	}

	
}

__host__ __device__ HashValue minExtractHeap(HashValue*A,  int size)
{
	//if (size < 1) is put outside
	
	HashValue min = A[0];
	A[0] = A[size - 1];
	minHeapify(A, 1, size);
	return min;
}

__host__ __device__ void buildMinHeap(HashValue* A, int size)//All heaps are min heap
{
	for (int i = (size / 2 - 1); i >= 0; i--)
	{
		minHeapify(A, i, size);
	}
}




void MQ_heapify(vector<HashValue>& MQ_host, vector<HashElem>& MQ_idx_host, size_t point_num, size_t MQ_size)
{
	for (int iter_morton = 0; iter_morton < MQ_idx_host.size(); iter_morton++)
	{
		int sz = MQ_idx_host[iter_morton].num-1;//minus one is necessary if you want sz to be index.
		int idx = MQ_idx_host[iter_morton].idx;
		if (sz == 0)continue;
		else
		{
			buildMinHeap(&MQ_host[idx], sz);
		}
	}
}
//Hashing these points to a hashvalue vector

__global__ void dev_MQ_heapify(HashValue* MQ_dev, HashElem* MQ_idx_dev,size_t MQ_size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < MQ_size)
	{
		int sz = MQ_idx_dev[idx].num;//minus one is necessary if you want sz to be index.
		int start_idx = MQ_idx_dev[idx].idx;
		if (sz == 0)return;
		else
		{
			buildMinHeap(&MQ_dev[start_idx], sz);
		}
	}
}

void hashingPoints2D(vector<Point2D>& points, vector<HashValue>& hash, size_t size, size_t grid_dimension)
{
	//calculate BBox of points

	BBox box;
	bBox(points, box, size);
	hash.resize(size);
	for (int i = 0; i < size; i++)
	{
		hash[i].morton = mortonHash2D(points[i], grid_dimension);
		hash[i].p = points[i];
	}

	//sort hash array
	sort(hash.begin(), hash.end());

	//the sorted hash array is the raw data of HashGrid.
	//the hash grid  need another array for indexing: HashIdx.

}
void hashingPoints2DWeighted(vector<HashValue>& points, vector<HashValue>& hash, size_t size, size_t grid_dimension)
{	
	hash.resize(size);
	for (int i = 0; i < size; i++)
	{
		hash[i].morton = mortonHash2D(points[i].p, grid_dimension);
		hash[i].p = points[i].p;
	}
	sort(hash.begin(), hash.end());
}


//this is going to generate a marker array
__host__ void generateMarkerArray(vector<int>& marker, vector<HashValue>& hash_grids, size_t size)
{
	for (int i = 0; i < size; i++)
	{
		if (i == 0)marker[i] = 1;
		else
		{
			if (hash_grids[i].morton != hash_grids[i - 1].morton)
			{
				marker[i] = 1;
			}
			else
			{
				marker[i] = 0;
			}
		}
	}
}





void hashIndexing(vector<HashValue>& hash_grids, vector<HashElem>& hash_idx, size_t size)
{
	

	//Prefix sum to get each morton code count;
	vector<int> marker;
	marker.resize(size);
	generateMarkerArray(marker, hash_grids, size);

	/*thrust::device_vector<int> offset_dev = marker;
	thrust::inclusive_scan(offset_dev.begin(), offset_dev.end(), offset_dev.begin());
	thrust::host_vector<int> offset_host(offset_dev.begin(), offset_dev.end());
*/
	//Now we have offset_host, we can do the indexing.

	//To make it easier, we make a count array.
	//each marker's index minus privious index(where marker is 1).
	vector<int> count;
	count.resize(size);
	int current_marker_offset = 0;
	int ct = 1;//counting
	for (int i = 0; i < size; i++)//iterate marker
	{
		if (marker[i] == 1)
		{
			count[current_marker_offset] = ct;
			ct = 1;
			current_marker_offset = i;//start counting
		}
		else
			ct++;
	}
	count[current_marker_offset] = ct;//solve the last = 0 bug

	//fill the hash_idx array
	for (int i = 0; i < size; i++)
	{
		if (marker[i] == 1)
		{
			int morton = hash_grids[i].morton;
			hash_idx[morton].idx = i;
			hash_idx[morton].num = count[i];
		}
		else
		{
			continue;
		}
	}
}


void matlabView(vector<Point2D> points, string filename)
{
	const string dir = "C:/Users/lunamu/Dropbox/MATLAB/";		//it's called matlabView for a reason
	ofstream file(dir + filename);
	for (int i = 0; i < points.size(); i++)
	{
		file << points[i].x << " " << points[i].y << endl;
	}
}

void axis(Point2D p, size_t grid_dim)
{
	int x_axis = (int)((p.x / 1.0) * (1<<grid_dim));
	int y_axis = (int)((p.y / 1.0) * (1<<grid_dim));
}

__host__ __device__ float distance2(Point2D p1, Point2D p2)
{
	return sqrtf((p1.x - p2.x) * (p1.x - p2.x) + (p1.y - p2.y) * (p1.y - p2.y));
}

//Original, buggy, host
//void assignOriginalWeighting(vector<HashValue>& hash_grids, vector<HashElem>& hash_idx, float radius, size_t grid_dim)
//{
//	for(int iter = 0; iter < hash_grids.size(); iter++)
//	{
//		//there's no need for a template radiusSearch function, just write the logic
//		//radiusSearch()
//		Point2D& current_point = hash_grids[iter].p;
//		int x_axis_start = (int)(((current_point-radius).x / 1.0) * (1<<grid_dim));
//		int y_axis_start = (int)(((current_point-radius).y / 1.0) * (1<<grid_dim));
//		int x_axis_end = (int)(((current_point+radius).x / 1.0) * (1<<grid_dim));
//		int y_axis_end = (int)(((current_point+radius).y / 1.0) * (1<<grid_dim));
//
//		for (int x = x_axis_start; x <= x_axis_end; x++)
//		{
//			for (int y = y_axis_start; y <= y_axis_end; y++)
//			{
//				
//				int morton = mortonHash2D_axis(x, y , grid_dim);
//				int offset = hash_idx[morton].idx;
//				for (int idx = 0; idx < hash_idx[morton].num; idx++)
//				{
//					float d = distance2(hash_grids[offset + idx].p, current_point);
//					//if (d < radius)current_point.w += d;
//					//test, only use the counting
//					if (d < radius)current_point.w += 1;
//				}
//			}
//		}
//
//
//	}
//}
#define EPS 0.00001
__global__ void devAssignOriginalWeighting(HashValue* hash_grids, HashElem* hash_idx, float radius, size_t grid_dim, size_t point_num)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < point_num)
	{
		Point2D& current_point = hash_grids[idx].p;
		Point2D& min_corner = (current_point - radius);
		Point2D& max_corner = (current_point + radius);
		int x_axis_start = (int)((min_corner.x / 1.0) * (1 << grid_dim));
		int y_axis_start = (int)((min_corner.y / 1.0) * (1 << grid_dim));
		int x_axis_end = (int)((max_corner.x / 1.0) * (1 << grid_dim));
		int y_axis_end = (int)((max_corner.y / 1.0) * (1 << grid_dim));

		for (int x = x_axis_start; x <= x_axis_end; x++)
		{
			for (int y = y_axis_start; y <= y_axis_end; y++)
			{
				int morton = mortonHash2D_axis(x, y, grid_dim);
				int offset = hash_idx[morton].idx;
				for (int idx_in_grid = 0; idx_in_grid < hash_idx[morton].num; idx_in_grid++)
				{
					float d = distance2(hash_grids[offset + idx_in_grid].p, current_point);
					//if (d < radius)current_point.w += d;
					//test, only use the counting
					if (d < EPS) continue;
					else if (d < radius)current_point.w += 1.0/d;
				}
			}
		}
	}
}
//only for test!
__global__ void OriginalIndexDevAssignOriginalWeighting(Point2D* points, HashValue* hash_grids, HashElem* hash_idx, float radius, size_t grid_dim, size_t point_num)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < point_num)
	{
		Point2D& current_point = points[idx];
		Point2D& min_corner = (current_point - radius);
		Point2D& max_corner = (current_point + radius);
		int x_axis_start = (int)((min_corner.x / 1.0) * (1 << grid_dim));
		int y_axis_start = (int)((min_corner.y / 1.0) * (1 << grid_dim));
		int x_axis_end = (int)((max_corner.x / 1.0) * (1 << grid_dim));
		int y_axis_end = (int)((max_corner.y / 1.0) * (1 << grid_dim));

		for (int x = x_axis_start; x <= x_axis_end; x++)
		{
			for (int y = y_axis_start; y <= y_axis_end; y++)
			{
				int morton = mortonHash2D_axis(x, y, grid_dim);
				int offset = hash_idx[morton].idx;
				for (int idx_in_grid = 0; idx_in_grid < hash_idx[morton].num; idx_in_grid++)
				{
					float d = distance2(hash_grids[offset + idx_in_grid].p, current_point);
					//if (d < radius)current_point.w += d;
					//test, only use the counting
					if (d < radius)current_point.w += 1;
				}
			}
		}
	}
}
__global__ void extractBatch(HashValue* dev_hash_grids, HashElem* dev_hash_idx, size_t morton_num, HashValue* results, size_t batchSize, int* randomized_grid_idx)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < batchSize)
	{
		int current_idx = randomized_grid_idx[idx];//map current randomized idx;
		int heap_idx = dev_hash_idx[current_idx].idx;
		int heap_size = dev_hash_idx[current_idx].num;
		if (heap_size < 1)
		{
			HashValue nil; nil.morton = 0; nil.p.x = 0; nil.p.y = 0; nil.p.w = 0;
			results[current_idx] = nil;
			return;
		}
		results[idx] = minExtractHeap(&dev_hash_grids[heap_idx], heap_size);

		//mega kernel, update in one kernel vs. update in different kernel;
		//currently update in different kernel;
		
		dev_hash_idx[current_idx].num -= 1;
	}
}


//To understand
//if radius is smaller than the radius recorded in w, then replace radius.
//but the heap is min heap, and we want to select w with large r, so radius is recorded in reciprocal form;
__host__ __device__ float weighting(Point2D target_point, Point2D query_point, float r)
{
	if (r == 0) return 0.0;//query point it self;
	else if (target_point.w - 0.0 < EPS) return 1.0 / r;
	else if (r < 1. / target_point.w) return 1.0 / r;
	else return target_point.w;
}
__global__ void updateWeight(HashValue* dev_hash_grids, HashElem* dev_hash_idx, HashValue* batch, float radius, size_t grid_dim, size_t batchSize)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < batchSize)
	{
		Point2D query_point = batch[idx].p;
		Point2D& min_corner = (query_point - radius);
		Point2D& max_corner = (query_point + radius);
		int x_axis_start = (int)((min_corner.x / 1.0) * (1 << grid_dim));
		int y_axis_start = (int)((min_corner.y / 1.0) * (1 << grid_dim));
		int x_axis_end = (int)((max_corner.x / 1.0) * (1 << grid_dim));
		int y_axis_end = (int)((max_corner.y / 1.0) * (1 << grid_dim));
		for (int x = x_axis_start; x <= x_axis_end; x++)
		{
			for (int y = y_axis_start; y <= y_axis_end; y++)
			{
				int morton = mortonHash2D_axis(x, y, grid_dim);
				int offset = dev_hash_idx[morton].idx;
				for (int idx_in_grid = 0; idx_in_grid < dev_hash_idx[morton].num; idx_in_grid++)
				{
					float d = distance2(dev_hash_grids[offset + idx_in_grid].p, query_point);
					//if (d < radius)current_point.w += d;
					//test, only use the counting
					if (d < EPS) continue;
					else if (d < radius)
					{
						//weighting strategy!
						dev_hash_grids[offset + idx_in_grid].p.w = weighting(dev_hash_grids[offset + idx_in_grid].p, query_point, d);

						//TODO::profile, immediately triger a decrease key (atomicly), or update the whole afterwards?
					}
				}
			}
		}

	}
}


void heapifytest(HashValue* MQ_dev, HashElem* MQ_idx_dev, size_t MQ_size)
{
	int sz = 49;//minus one is necessary if you want sz to be index.
	int idx = 0;
	if (sz == 0)return;
	else
	{
		buildMinHeap(&MQ_dev[idx], sz);
	}
}

#define BIGF 10000.0;
__global__ void resetWeighting(HashValue* dev_hash_grids, size_t size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		dev_hash_grids[idx].p.w = 0.0;
	}
}
int main()
{
	//important parameters
	int grid_dim = 7;
	int point_num = 1000000;
	float radius = 0.001;



	int morton_num = (1 << grid_dim) * (1 << grid_dim);
	vector<Point2D> points;
	vector<HashValue> hash_grids;
	vector<HashElem> hash_idx;  //size is the maximum size of a certain dimension;
	hash_idx.resize(morton_num);
	for (int i = 0; i < morton_num; i++){ hash_idx[i].idx = 0; hash_idx[i].num = 0; }//init hash_idx array

	//generate random point cloud 2d;
	generateRandomPointCloud(points, point_num);
	////view these points in matlab;
	//matlabView(points,"results");
	//make hash_grids from the random points.

	//time test
#ifdef TEST
	for (int i = 10000; i <= 500000; i += 10000)
	{
		generateRandomPointCloud(points, i);
		int start_s = clock();
		hashingPoints2D(points, hash_grids, i, grid_dim);
		//make hash index (used for searching)

		hashIndexing(hash_grids, hash_idx, i);
		int stop_s = clock();

		cout << i << " " << (stop_s - start_s) / double(CLOCKS_PER_SEC) * 1000.0 << endl;
	}
#endif

	hashingPoints2D(points, hash_grids, point_num, grid_dim);
	//make hash index (used for searching)

	hashIndexing(hash_grids, hash_idx, point_num);
	

	//Now, all important index are built, move to GPU;
	HashValue* dev_hash_grids = 0;
	HashElem* dev_hash_idx = 0;
	Point2D* dev_points = 0;//only for test.
	GPUCHECK(hipMalloc((void**)&dev_hash_grids, point_num * sizeof(HashValue)));
	GPUCHECK(hipMalloc((void**)&dev_hash_idx, morton_num * sizeof(HashElem)));
	GPUCHECK(hipMalloc((void**)&dev_points, point_num * sizeof(Point2D)));
	//copy to GPU
	GPUCHECK(hipMemcpy(dev_hash_grids, &hash_grids[0], point_num * sizeof(HashValue), hipMemcpyHostToDevice));
	GPUCHECK(hipMemcpy(dev_hash_idx, &hash_idx[0], morton_num * sizeof(HashElem), hipMemcpyHostToDevice));
	GPUCHECK(hipMemcpy(dev_points, &points[0], point_num * sizeof(Point2D), hipMemcpyHostToDevice));


	//generate original weighting 
	//1. assign each weighting as num of points within radius.
	//host version
	//assignOriginalWeighting(hash_grids, hash_idx, radius, grid_dim);

	int threadsPerBlock = 256;
	int numBlocks = (point_num + threadsPerBlock - 1) / threadsPerBlock;

	//This is only going to compare with the results from MATLAB's radius search.
	//OriginalIndexDevAssignOriginalWeighting << <numBlocks, threadsPerBlock >> >(dev_points, dev_hash_grids, dev_hash_idx, radius, grid_dim, point_num);

	//dev weighting assignment.
	devAssignOriginalWeighting << <numBlocks, threadsPerBlock >> >(dev_hash_grids, dev_hash_idx, radius, grid_dim, point_num);
	//viewGPUArray<HashValue>(dev_hash_grids, point_num, "bc_dev_hash_grids");
	//viewGPUArray<HashValue>(dev_hash_grids, point_num, "dev_hash_grids");
	//heapify
	numBlocks = (morton_num + threadsPerBlock - 1) / threadsPerBlock;
	dev_MQ_heapify << <numBlocks, threadsPerBlock >> >(dev_hash_grids, dev_hash_idx,  morton_num);

	//HashValue* host_hash_grids = (HashValue*)malloc(sizeof(HashValue) * point_num);
	//HashElem* host_hash_idx = (HashElem*)malloc(sizeof(HashValue) * point_num);
	//vector<HashValue> host_hash_grids(point_num);
	//vector<HashElem> host_hash_idx(morton_num);
	//GPUCHECK(hipMemcpy(&host_hash_grids[0], dev_hash_grids, sizeof(HashValue) * point_num, hipMemcpyDeviceToHost));
	//GPUCHECK(hipMemcpy(&host_hash_idx[0], dev_hash_idx, sizeof(HashElem) * morton_num, hipMemcpyDeviceToHost));


	//heapifytest(&host_hash_grids[0], &host_hash_idx[0], morton_num);

	//viewGPUArray<HashValue>(dev_hash_grids, point_num, "dev_hash_grids");
	//viewGPUArray<HashElem>(dev_hash_idx, morton_num, "dev_hash_idx");
	vector<int> randomized_index_array;

	int batchSize = morton_num / 4;
	int desiredNum = batchSize * 16;

	int* dev_randomized_index_array;
	randomized_index_array.resize(batchSize);
	for (int i = 0; i < randomized_index_array.size(); i++)
	{
		//TODO:randomize
		randomized_index_array[i] = i;
	}

	HashValue* desiredResults;
	GPUCHECK(hipMalloc((void**)&dev_randomized_index_array, sizeof(int) * batchSize));
	GPUCHECK(hipMalloc((void**)&desiredResults, sizeof(HashValue) * desiredNum));
	GPUCHECK(hipMemcpy(dev_randomized_index_array, &randomized_index_array[0], sizeof(int) * batchSize, hipMemcpyHostToDevice))
	
	int threadsPerBlock_batch = 64;
	int numBlocks_batch = (batchSize + threadsPerBlock_batch - 1) / threadsPerBlock_batch;
	//extract those calculated with radius search
	

	viewGPUArray<HashElem>(dev_hash_idx, morton_num, "bc_dev_hash_idx");
	
	extractBatch << <numBlocks_batch, threadsPerBlock_batch >> >(dev_hash_grids, dev_hash_idx, morton_num, desiredResults, batchSize, dev_randomized_index_array);

	viewGPUArrayMat<HashValue>(desiredResults, batchSize, "batch");
	//reset weighting
	int threadsPerBlock_reset_weighting = 256;
	int numBlocks_reset_weighting = (point_num + threadsPerBlock_reset_weighting - 1) / threadsPerBlock_reset_weighting;
	resetWeighting << <threadsPerBlock_reset_weighting, numBlocks_reset_weighting >> >(dev_hash_grids, point_num);


	//update weighting
	int threadsPerBlock_updateWeighting = 64;
	int numBlocks_updateWeighting = (batchSize + threadsPerBlock_updateWeighting - 1) / threadsPerBlock_updateWeighting;
	int offset = 0;//offset of this batch
	updateWeight << <threadsPerBlock_updateWeighting, numBlocks_updateWeighting >> >(dev_hash_grids, dev_hash_idx, desiredResults, radius, grid_dim, batchSize);


	
	for (int batch_idx = 1; batch_idx < desiredNum/batchSize; batch_idx++)
	{
		int tpb_heapify = 256;
		int nb_heapify = (point_num + tpb_heapify - 1) / tpb_heapify;
		dev_MQ_heapify << <tpb_heapify, nb_heapify >> >(dev_hash_grids, dev_hash_idx, morton_num);
		
		int tpb_extract = 64;
		int nb_extract = (batchSize + tpb_extract - 1) / tpb_extract;
		extractBatch << <tpb_extract, nb_extract >> >(dev_hash_grids, dev_hash_idx, morton_num, desiredResults+batchSize*batch_idx, batchSize, dev_randomized_index_array);

		int tpb_update = 64;
		int nb_update = (batchSize + tpb_update - 1) / threadsPerBlock_updateWeighting;
		updateWeight << <threadsPerBlock_updateWeighting, numBlocks_updateWeighting >> >(dev_hash_grids, dev_hash_idx, desiredResults + batch_idx * batchSize, radius, grid_dim, batchSize);

	}

	viewGPUArray<HashValue>(desiredResults, desiredNum, "batch");
	viewGPUArrayMat<HashValue>(desiredResults, desiredNum, "batch");
	
	GPUCHECK(hipFree(dev_hash_grids));
	GPUCHECK(hipFree(dev_hash_idx));
	GPUCHECK(hipFree(dev_points));
	GPUCHECK(hipFree(desiredResults));
	GPUCHECK(hipFree(dev_randomized_index_array));

	/*
	GPUCHECK(hipFree(MQ_dev));
	GPUCHECK(hipFree(MQ_idx_dev));
*/

    return 0;
}


